#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "common/common.h"

#include <stdio.h>
#include <stdint.h>
#include <sqlite3.h>

#include "struct.h"
#include "makros.h"

#include "histogram.h"
#include "slice2chunk.h"
#include "kinetics.h"

#include "point_square.h"
#include "device_info.h"



// plot_raw_filtered print all raw data between left_chunk_idx and right_chunk_idx with outliers excluded.
// used for creating plots to R to debug and analysis. to run from BASH and inside main
// Must first run:
// 1) kinetics2record - the kinetics file to datastructure magtable
// 2) gps2chunk_record  - Creates chunks pointing into mag_table
// 3) histogram       - cut off outliers and mark it in mag_table[idx].outlier
void plot_raw_filtered(chunk_record *chunk_table, int *chunk_len, mag_record *mag_table, int *mag_len, int left_chunk_idx, int right_chunk_idx) {
    short mxt;
    short myt;

    puts("mxt, myt");

    for (int mag_idx = chunk_table[left_chunk_idx].left_mag_idx; mag_idx <= chunk_table[right_chunk_idx].right_mag_idx; mag_idx++) {
        mxt = mag_table[mag_idx].mxt;
        myt = mag_table[mag_idx].myt;

        if (!mag_table[mag_idx].disable) {
            printf("%i,%i\n", mxt, myt);
        }
    }
  }



int main(int argc, char *argv[]) {

    fprintf(stderr,"\n\n                               *** OSAN POSITIONING 2017 v0.01 ***\n\n");

    print_device_info();    // Print out all relevant CUDA device information

    if (argc != 2) {
       fprintf(stderr,"Usage:\n");
       fprintf(stderr,"normalize <database>\n\n");
       exit(1);
    }

    // set up CUDA device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    char buffer_Z[100];  // string buffer

    int mag_len;
    mag_record *mag_table = NULL;

    int chunk_len;
    chunk_record *chunk_table = NULL;


    // skriv ut en text här hur man refererar till programmet om man publicerar
    // vetenskapliga resultat. OSAN POSITIONING; H-H. Fuxelius

    sprintf(buffer_Z,"%s",argv[1]);   // *.sqlite3

    // Reads in magnetometer data from database (table kinetics) to magtable and returns
    // table length kinetics_len
    kinetics2record(buffer_Z, &mag_table, &mag_len);

    // Creates an chunk_table which is a partitioning of mxt, myt of a chunk_size
    // chunk_table[].left_mag_idx and chunk_table[].right_mag_idx points out each chunk border
    // These chunk partition the entire mag_table
    int chunk_size = 1024;           // Should be a multiple of BLOCK_SIZE (now set to 256); CUDA stuff
    slice2chunk_record(&chunk_table, &chunk_len, mag_table, mag_len, chunk_size);


    #ifdef DEBUG_INFO_1
        // Proves that the pointers are correct in chunk_table
        puts("seqid | seqid | mag_idx | mag_idx | seqid | seqid");

        int left_idx, right_idx;
        for (int rec_cnt=0; rec_cnt<chunk_len; rec_cnt++) {
            printf("++-> %u | %u | %u |",rec_cnt, chunk_table[rec_cnt].left_seq_id, chunk_table[rec_cnt].right_seq_id);
            printf(" %u | %u | ", chunk_table[rec_cnt].left_mag_idx, chunk_table[rec_cnt].right_mag_idx);

            left_idx = chunk_table[rec_cnt].left_mag_idx;
            right_idx = chunk_table[rec_cnt].right_mag_idx;

            printf("%u | %u \n", mag_table[left_idx].seq_id, mag_table[right_idx].seq_id);

        }
    #endif


    // Run histogram on each chunk, and store its results in chunk_table
    int bin   = 5;
    int range = 100; // => (-500,+500)
    int cut_off = 5;
    for (int chunk_idx=0; chunk_idx<chunk_len; chunk_idx++) {
        histogram(chunk_table, &chunk_len, mag_table, &mag_len, chunk_idx, bin, range, cut_off);
    }

    // print out the info in all chunks
    #ifdef DEBUG_INFO_1
        for (int chunk_idx=0; chunk_idx<chunk_len; chunk_idx++) {
            printf("chunk_idx %i\n", chunk_idx);
            printf("left_mag_idx %i\n", chunk_table[chunk_idx].left_mag_idx);
            printf("right_mag_idx %i\n", chunk_table[chunk_idx].right_mag_idx);
            printf("x0 %f\n", chunk_table[chunk_idx].x0);
            printf("y0 %f\n", chunk_table[chunk_idx].y0);
            printf("scale_r %f\n", chunk_table[chunk_idx].scale_r);
            printf("scale_y_axis %f\n", chunk_table[chunk_idx].scale_y_axis);
            printf("theta %f\n", chunk_table[chunk_idx].theta);
            printf("disable %i\n\n", chunk_table[chunk_idx].disable);
        }
    #endif
    //--------------------------------------------------------------------------

    int chunk_idx = 0;

    // tested model
    short mxt      =  200;
    short myt      =  -30;
    float x0       =   16;
    float y0       =  -124;
    float scale_r  = 0.0041;
    float scale_y  = 1.045; // 1.045
    float rotate   = 0.0;

    float normalized_x;  // Return value
    float normalized_y;  // Return value
    float quad_error;    // Return value

    for (int mag_idx = chunk_table[chunk_idx].left_mag_idx; mag_idx <= chunk_table[chunk_idx].right_mag_idx; mag_idx++) {
        mxt = mag_table[mag_idx].mxt;
        myt = mag_table[mag_idx].myt;

        if (!mag_table[mag_idx].disable) {
            point_square(mxt, myt, x0, y0, scale_r, scale_y, rotate, &normalized_x, &normalized_y, &quad_error);
            //printf("%f,%f\n", normalized_x, normalized_y);
        }
    }


      //point_square_GPU(&chunk_table, chunk_len, &mag_table, mag_len, chunk_size);


    // malloc device global memory
    mag_record *d_mag_table;
    size_t mag_bytes = mag_len * sizeof(mag_record);
    CHECK(hipMalloc((void **)&d_mag_table, mag_bytes));
    CHECK(hipMemcpy(d_mag_table, mag_table, mag_bytes, hipMemcpyHostToDevice));

    chunk_record *d_chunk_table;
    size_t chunk_bytes = chunk_len * sizeof(chunk_record);
    CHECK(hipMalloc((void **)&d_chunk_table, chunk_bytes));
    CHECK(hipMemcpy(d_chunk_table, chunk_table, chunk_bytes, hipMemcpyHostToDevice));


    // invoke kernel at host side
    int dimx = BLOCK_SIZE; // < 1024
    dim3 block(dimx, 1);
    dim3 grid(mag_len / block.x + 1, 1);
    //dim3 grid(800, 1);

    //point_square_GPU(&chunk_table, chunk_len, &mag_table, mag_len, chunk_size);

    point_square_GPU<<<grid, block>>>(d_chunk_table, chunk_len, d_mag_table, mag_len, chunk_size);

    CHECK(hipDeviceSynchronize());

    CHECK(hipGetLastError());

    CHECK(hipMemcpy(mag_table, d_mag_table, mag_bytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(chunk_table, d_chunk_table, chunk_bytes, hipMemcpyDeviceToHost));

    // free device global memory
    CHECK(hipFree(d_mag_table));
    CHECK(hipFree(d_chunk_table));

    // reset device
    CHECK(hipDeviceReset());

    free(mag_table);
    free(chunk_table);

    return 0;

}
