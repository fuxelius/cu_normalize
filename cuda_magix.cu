#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include "common/common.h"
#include <stdio.h>
#include <stdint.h>
#include "math.h"
#include "struct.h"
#include "makros.h"

// ====================================================== DATA STRUCTURES ==================================================================
// error_table[chunk_idx*CHUNK_SIZE + threadIdx.x] ??
// error_table[idx]
__device__ float error_table[META_SIZE * CHUNK_SIZE]; // meta_chunk_size * chunk_size = 102400 threads (410 kbyte)


// ========================================================== RANDOMIZE ====================================================================
__device__ float randomize(void) { // Return a random number between 0-1, make a simple implementation hmm use CURAND
    return 1;
}


// ======================================================== SUM A VECTOR ===================================================================
// this is probably very efficient ... if running on 100 processors in paralell ... and only 1024 loops ;)
__device__ float sum_vector(int chunk_idx, int chunk_size) {
    int off_set = chunk_idx*chunk_size;
    float sum = 0;

    for (int idx=off_set; idx < off_set+chunk_size; idx++) {
        sum = sum + error_table[idx];
    }

    printf("Sum vector: chunk_idx %i (%f)\n", chunk_idx, sum);

    return sqrtf(sum);
}


// ======================================================== POINT SQUARE ==================================================================
//CUDA implementation, hold the number of (mxt,myt) pairs <= 1024 to fit on a single SM, important for calculating the sum??!!
__global__ void point_square_GPU(chunk_record *chunk_table, int chunk_len,
                                     mag_record *mag_table, int mag_len,
                                                            int chunk_size) {

    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int chunk_idx = idx / chunk_size; // whole number
    int mag_idx = idx;

    printf("Point Square idx=%i chunk_idx=%i\n", idx, chunk_idx);

    if ((idx < mag_len) && !(mag_table[mag_idx].disable)) {
        // mag_table
        short mxt = mag_table[mag_idx].mxt;
        short myt = mag_table[mag_idx].myt;

        // // chunk_table
        // float x0            = chunk_table[chunk_idx].x0;
        // float y0            = chunk_table[chunk_idx].y0;
        // float scale_r       = chunk_table[chunk_idx].scale_r;
        // float scale_y_axis  = chunk_table[chunk_idx].scale_y_axis;
        // float theta         = chunk_table[chunk_idx].theta;

        // chunk_table; ga_uppsala2
        float x0            = 16;
        float y0            = -124;
        float scale_r       = 0.0041;
        float scale_y_axis  = 1.045;
        float theta         = 0;

        //printf("raw,%f,%f\n", mxt, myt);

        float trans_x = (mxt - x0);  // move plane to set origo to in middle of ellipse
        float trans_y = (myt - y0);  // move plane to set origo to in middle of ellipse

        //printf("trans,%f,%f\n", trans_x, trans_y);

        // Vector is individual for each rotated pair (mx,my)
        // unless they are situated on a perfect circle which
        // measured values are NOT!! (otherwise v would be r)

        // https://en.wikipedia.org/wiki/Rotation_(mathematics)
        // https://en.wikipedia.org/wiki/C_mathematical_functions

        float rotate_x = trans_x * cosf(theta) - trans_y * sinf(theta);  // Rotate theta degrees
        float rotate_y = trans_x * sinf(theta) + trans_y * cosf(theta);  // Rotate theta degrees

        //printf("rotate,%f,%f\n", rotate_x, rotate_y);

        float scale_x = rotate_x;
        float scale_y = rotate_y * scale_y_axis;                         // Scale y-axis to make the ellips to a cicle

        //printf("scale,%f,%f\n", scale_x, scale_y); // För R-plot

        rotate_x = scale_x * cosf(-theta) - scale_y * sinf(-theta);      // Rotate -theta degrees (back)
        rotate_y = scale_x * sinf(-theta) + scale_y * cosf(-theta);      // Rotate -theta degrees (back)

        //printf("rotate back ,%f,%f\n", rotate_x, rotate_y);

        float normalized_x = rotate_x * scale_r;                        // Returns x,y normalized to unit circle
        float normalized_y = rotate_y * scale_r;

        //printf("normalized ,%f,%f\n", normalized_x, normalized_y);

        //printf("cuda,%f,%f\n", scale_x, scale_y);

        float quad_error = powf(sqrtf(powf(normalized_x,2) + powf(normalized_y,2)) - 1,2); // Returns square error from unity cicle

        //printf("quad_error,%f\n", quad_error);

        error_table[idx] = quad_error;

        // Write back result
        //mag_table[mag_idx].normalized_x = normalized_x;
        //mag_table[mag_idx].normalized_y = normalized_y;
        //mag_table[mag_idx].quad_error   = quad_error;
        //result_table[mag_idx].mfv = normalized_x;
        //result_table[mag_idx].rho = normalized_y;

        //printf("first test x=%i, y=%i, x+y=%i\n", 3, 4, first_test(3,4));

     }
}


// ======================================================== PARENT LAUNCH =================================================================
__global__ void parent_launch(chunk_record *chunk_table, int chunk_len,
                                  mag_record *mag_table, int mag_len,
                                  meta_record *meta_table, int meta_len,
                                                         int chunk_size) {

    printf("Parent Launch: %i\n", threadIdx.x);

    int meta_idx = 0;
    int left_chunk_idx = 0;
    int right_chunk_idx = 0;

    left_chunk_idx  = meta_table[meta_idx].left_chunk_idx;
    right_chunk_idx = meta_table[meta_idx].right_chunk_idx;

    point_square_GPU<<<1, ((right_chunk_idx-left_chunk_idx+1) * chunk_size)>>>(chunk_table, chunk_len, mag_table, mag_len, CHUNK_SIZE);

    hipDeviceSynchronize();

    int chunk_idx = 0;

    float sum = sum_vector(chunk_idx, CHUNK_SIZE);

    hipDeviceSynchronize();

    printf("LSQ=%f for chunk_idx=%i\n", sum, chunk_idx);
}


// ========================================================= HOST LAUNCH ==================================================================
void host_launch(chunk_record *chunk_table, int chunk_len,
                                  mag_record *mag_table, int mag_len,
                                  meta_record *meta_table, int meta_len,
                                                         int chunk_size) {
    printf("Host Launch:\n");

    parent_launch<<<1,1>>>(chunk_table, chunk_len, mag_table, mag_len, meta_table, meta_len, chunk_size);
}
